#include "hip/hip_runtime.h"
#include <rclcpp/rclcpp.hpp>
#include <std_msgs/msg/float32_multi_array.hpp>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void addConstant(float* data, float constant, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) data[idx] += constant;
}

class CudaNode : public rclcpp::Node {
public:
  CudaNode() : Node("cuda_node") {
    sub_ = create_subscription<std_msgs::msg::Float32MultiArray>(
      "/ros2/input", 10,
      std::bind(&CudaNode::callback, this, std::placeholders::_1));
    pub_ = create_publisher<std_msgs::msg::Float32MultiArray>(
      "/ros2/output", 10);
  }

private:
  void callback(const std_msgs::msg::Float32MultiArray::SharedPtr msg) {
    size_t N = msg->data.size();
    size_t bytes = N * sizeof(float);

    // 1) Allocate device memory *every callback*
    float* d_data;
    hipMalloc(&d_data, bytes);

    // 2) Time from first copy to final copy
    auto start = std::chrono::high_resolution_clock::now();

    // 3) Host→Device
    hipMemcpy(d_data, msg->data.data(), bytes, hipMemcpyHostToDevice);

    // 4) Kernel launch
    int threads = 256;
    int blocks  = (N + threads - 1) / threads;
    addConstant<<<blocks, threads>>>(d_data, 5.0f, N);
    hipDeviceSynchronize();

    // 5) Device→Host
    std::vector<float> output(N);
    hipMemcpy(output.data(), d_data, bytes, hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    auto ms  = std::chrono::duration<double, std::milli>(end - start).count();
    RCLCPP_INFO(get_logger(), "GPU processing took %.2f ms", ms);

    hipFree(d_data);

    // 6) Publish result
    auto out_msg = std_msgs::msg::Float32MultiArray();
    out_msg.data = std::move(output);
    pub_->publish(std::move(out_msg));
  }

  rclcpp::Subscription<std_msgs::msg::Float32MultiArray>::SharedPtr sub_;
  rclcpp::Publisher<std_msgs::msg::Float32MultiArray>::SharedPtr pub_;
};

int main(int argc, char** argv) {
  rclcpp::init(argc, argv);
  rclcpp::spin(std::make_shared<CudaNode>());
  rclcpp::shutdown();
  return 0;
}