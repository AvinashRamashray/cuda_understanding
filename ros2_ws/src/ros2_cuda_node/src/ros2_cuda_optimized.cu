#include "hip/hip_runtime.h"
#include <rclcpp/rclcpp.hpp>
#include <std_msgs/msg/float32_multi_array.hpp>
#include <hip/hip_runtime.h>

__global__ void addConstantKernel(float* data, float constant, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) data[idx] += constant;
}

class OptimizedCudaNode : public rclcpp::Node {
public:
  OptimizedCudaNode()
  : Node("optimized_cuda_node"),
    N_(1 << 20),
    bytes_(N_ * sizeof(float)),
    constant_(5.0f)
  {
    // 1) Preallocate pinned host buffers (faster transfers)
    hipHostMalloc(&h_input_,  bytes_);
    hipHostMalloc(&h_output_, bytes_);

    // 2) Preallocate device buffer once
    hipMalloc(&d_data_, bytes_);

    // 3) Create a single CUDA stream
    hipStreamCreate(&stream_);

    // 4) Create CUDA events for timing
    hipEventCreate(&evt_h2d_);
    hipEventCreate(&evt_kernel_);
    hipEventCreate(&evt_d2h_);

    // 5) ROS2 interfaces
    sub_ = create_subscription<std_msgs::msg::Float32MultiArray>(
      "/ros2/input", 1,
      std::bind(&OptimizedCudaNode::callback, this, std::placeholders::_1));
    pub_ = create_publisher<std_msgs::msg::Float32MultiArray>(
      "/ros2/output", 1);
  }

  ~OptimizedCudaNode() {
    hipHostFree(h_input_);
    hipHostFree(h_output_);
    hipFree(d_data_);
    hipStreamDestroy(stream_);
    hipEventDestroy(evt_h2d_);
    hipEventDestroy(evt_kernel_);
    hipEventDestroy(evt_d2h_);
  }

private:
  void callback(const std_msgs::msg::Float32MultiArray::SharedPtr msg) {
    // Copy into pinned buffer
    memcpy(h_input_, msg->data.data(), bytes_);

    // H2D
    hipEventRecord(evt_h2d_, stream_);
    hipMemcpyAsync(d_data_, h_input_, bytes_, hipMemcpyHostToDevice, stream_);

    // Kernel
    hipEventRecord(evt_kernel_, stream_);
    const int threads = 256;
    const int blocks  = (N_ + threads - 1) / threads;
    addConstantKernel<<<blocks, threads, 0, stream_>>>(d_data_, constant_, N_);

    // D2H
    hipEventRecord(evt_d2h_, stream_);
    hipMemcpyAsync(h_output_, d_data_, bytes_, hipMemcpyDeviceToHost, stream_);

    // Wait once
    hipStreamSynchronize(stream_);

    // Read timings
    float t_h2d = 0.f, t_kern = 0.f, t_d2h = 0.f;
    hipEventElapsedTime(&t_h2d,  evt_h2d_,  evt_kernel_);
    hipEventElapsedTime(&t_kern, evt_kernel_, evt_d2h_);
    hipEventElapsedTime(&t_d2h,  evt_d2h_,  evt_d2h_);
    RCLCPP_INFO(get_logger(),
      "Timings (ms): H2D=%.2f  Kernel=%.2f  D2H=%.2f",
      t_h2d, t_kern, t_d2h);

    // Publish result
    auto out = std_msgs::msg::Float32MultiArray();
    out.data.assign(h_output_, h_output_ + N_);
    pub_->publish(std::move(out));
  }

  // Members
  size_t N_, bytes_;
  float  constant_;
  float *h_input_, *h_output_, *d_data_;
  hipStream_t stream_;
  hipEvent_t   evt_h2d_, evt_kernel_, evt_d2h_;
  rclcpp::Subscription<std_msgs::msg::Float32MultiArray>::SharedPtr sub_;
  rclcpp::Publisher<std_msgs::msg::Float32MultiArray>::SharedPtr pub_;
};

int main(int argc, char** argv) {
  rclcpp::init(argc, argv);
  rclcpp::spin(std::make_shared<OptimizedCudaNode>());
  rclcpp::shutdown();
  return 0;
}